#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "routinesGPU.h"
#include "png_io.h"

#define DEG2RAD 0.017453f

#define BLOCK_SIZE 16

__global__ void init_cos_sin_table_kernel(float *sin_table, float *cos_table, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        sin_table[i] = sinf(i * DEG2RAD);
        cos_table[i] = cosf(i * DEG2RAD);
    }
}

__global__ void image_RGB2BW_kernel(uint8_t *image_in, uint8_t *image_out, int height, int width)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    float R, G, B;

    if (i < height && j < width)
    {
        R = (float)(image_in[3 * (i * width + j)    ]);
        G = (float)(image_in[3 * (i * width + j) + 1]);
        B = (float)(image_in[3 * (i * width + j) + 2]);

        image_out[i * width + j] = (uint8_t)(0.2989 * R + 0.5870 * G + 0.1140 * B);
    }
}

__global__ void noiseReduction_kernel(uint8_t *im, float *NR, int height, int width)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= 2 && i < height - 2 && j >= 2 && j < width - 2)
	{
		// Noise reduction
		NR[i * width + j] = (2.0 * im[(i - 2) * width + (j - 2)] + 4.0 * im[(i - 2) * width + (j - 1)] + 5.0 * im[(i - 2) * width + (j)] + 4.0 * im[(i - 2) * width + (j + 1)] + 2.0 * im[(i - 2) * width + (j + 2)] + 4.0 * im[(i - 1) * width + (j - 2)] + 9.0 * im[(i - 1) * width + (j - 1)] + 12.0 * im[(i - 1) * width + (j)] + 9.0 * im[(i - 1) * width + (j + 1)] + 4.0 * im[(i - 1) * width + (j + 2)] + 5.0 * im[(i)*width + (j - 2)] + 12.0 * im[(i)*width + (j - 1)] + 15.0 * im[(i)*width + (j)] + 12.0 * im[(i)*width + (j + 1)] + 5.0 * im[(i)*width + (j + 2)] + 4.0 * im[(i + 1) * width + (j - 2)] + 9.0 * im[(i + 1) * width + (j - 1)] + 12.0 * im[(i + 1) * width + (j)] + 9.0 * im[(i + 1) * width + (j + 1)] + 4.0 * im[(i + 1) * width + (j + 2)] + 2.0 * im[(i + 2) * width + (j - 2)] + 4.0 * im[(i + 2) * width + (j - 1)] + 5.0 * im[(i + 2) * width + (j)] + 4.0 * im[(i + 2) * width + (j + 1)] + 2.0 * im[(i + 2) * width + (j + 2)]) / 159.0;
	}
}

__global__ void gradient_kernel(float *NR, float *G, float *phi, float *Gx, float *Gy,
	int height, int width)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	float PI = 3.141593;

	if (i >= 2 && i < height - 2 && j >= 2 && j < width - 2)
	{
		// Intensity gradient of the image
		Gx[i * width + j] =
			(1.0 * NR[(i - 2) * width + (j - 2)] + 2.0 * NR[(i - 2) * width + (j - 1)] + (-2.0) * NR[(i - 2) * width + (j + 1)] + (-1.0) * NR[(i - 2) * width + (j + 2)] + 4.0 * NR[(i - 1) * width + (j - 2)] + 8.0 * NR[(i - 1) * width + (j - 1)] + (-8.0) * NR[(i - 1) * width + (j + 1)] + (-4.0) * NR[(i - 1) * width + (j + 2)] + 6.0 * NR[(i)*width + (j - 2)] + 12.0 * NR[(i)*width + (j - 1)] + (-12.0) * NR[(i)*width + (j + 1)] + (-6.0) * NR[(i)*width + (j + 2)] + 4.0 * NR[(i + 1) * width + (j - 2)] + 8.0 * NR[(i + 1) * width + (j - 1)] + (-8.0) * NR[(i + 1) * width + (j + 1)] + (-4.0) * NR[(i + 1) * width + (j + 2)] + 1.0 * NR[(i + 2) * width + (j - 2)] + 2.0 * NR[(i + 2) * width + (j - 1)] + (-2.0) * NR[(i + 2) * width + (j + 1)] + (-1.0) * NR[(i + 2) * width + (j + 2)]);

		Gy[i * width + j] =
			((-1.0) * NR[(i - 2) * width + (j - 2)] + (-4.0) * NR[(i - 2) * width + (j - 1)] + (-6.0) * NR[(i - 2) * width + (j)] + (-4.0) * NR[(i - 2) * width + (j + 1)] + (-1.0) * NR[(i - 2) * width + (j + 2)] + (-2.0) * NR[(i - 1) * width + (j - 2)] + (-8.0) * NR[(i - 1) * width + (j - 1)] + (-12.0) * NR[(i - 1) * width + (j)] + (-8.0) * NR[(i - 1) * width + (j + 1)] + (-2.0) * NR[(i - 1) * width + (j + 2)] + 2.0 * NR[(i + 1) * width + (j - 2)] + 8.0 * NR[(i + 1) * width + (j - 1)] + 12.0 * NR[(i + 1) * width + (j)] + 8.0 * NR[(i + 1) * width + (j + 1)] + 2.0 * NR[(i + 1) * width + (j + 2)] + 1.0 * NR[(i + 2) * width + (j - 2)] + 4.0 * NR[(i + 2) * width + (j - 1)] + 6.0 * NR[(i + 2) * width + (j)] + 4.0 * NR[(i + 2) * width + (j + 1)] + 1.0 * NR[(i + 2) * width + (j + 2)]);

		G[i * width + j] = sqrtf((Gx[i * width + j] * Gx[i * width + j]) + (Gy[i * width + j] * Gy[i * width + j])); // G = √Gx²+Gy²
		phi[i * width + j] = atan2f(fabs(Gy[i * width + j]), fabs(Gx[i * width + j]));
		
		if (fabs(phi[i * width + j]) <= PI / 8)
			phi[i * width + j] = 0;
		else if (fabs(phi[i * width + j]) <= 3 * (PI / 8))
			phi[i * width + j] = 45;
		else if (fabs(phi[i * width + j]) <= 5 * (PI / 8))
			phi[i * width + j] = 90;
		else if (fabs(phi[i * width + j]) <= 7 * (PI / 8))
			phi[i * width + j] = 135;
		else
			phi[i * width + j] = 0;
	} 
}

__global__ void edge_kernel(float *G, float *phi, uint8_t *pedge, int height, int width)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i >= 3 && i < height - 3 && j >= 3 && j < width - 3)
    {
        // Edge
        pedge[i * width + j] = 0;
        if (phi[i * width + j] == 0)
        {
            if (G[i * width + j] > G[i * width + j + 1] && G[i * width + j] > G[i * width + j - 1]) // edge is in N-S
                pedge[i * width + j] = 1;
        }
        else if (phi[i * width + j] == 45)
        {
            if (G[i * width + j] > G[(i + 1) * width + j + 1] && G[i * width + j] > G[(i - 1) * width + j - 1]) // edge is in NW-SE
                pedge[i * width + j] = 1;
        }
        else if (phi[i * width + j] == 90)
        {
            if (G[i * width + j] > G[(i + 1) * width + j] && G[i * width + j] > G[(i - 1) * width + j]) // edge is in E-W
                pedge[i * width + j] = 1;
        }
        else if (phi[i * width + j] == 135)
        {
            if (G[i * width + j] > G[(i + 1) * width + j - 1] && G[i * width + j] > G[(i - 1) * width + j + 1]) // edge is in NE-SW
                pedge[i * width + j] = 1;
        }
    }
}

__global__ void thresholding_kernel(uint8_t *im, uint8_t *image_out,
	float *G, uint8_t *pedge, float level, int height, int width)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= 3 && i < height - 3 && j >= 3 && j < width - 3)
	{
		// Hysteresis Thresholding
        float lowthres = level / 2;
        float hithres = 2 * (level);

        image_out[i * width + j] = 0;
        if (G[i * width + j] > hithres && pedge[i * width + j])
            image_out[i * width + j] = 255;
        else if (pedge[i * width + j] && G[i * width + j] >= lowthres && G[i * width + j] < hithres)
            // check neighbours 3x3
            for (int ii = -1; ii <= 1; ii++)
                for (int jj = -1; jj <= 1; jj++)
                    if (G[(i + ii) * width + j + jj] > hithres)
                        image_out[i * width + j] = 255;
	}
}

__global__ void hough_kernel(uint8_t *im, int width, int height, uint32_t *accumulators, int accu_width, int accu_height,
	float *sin_table, float *cos_table, float hough_h, float center_x, float center_y)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < height && j < width)
	{
		if (im[(i * width) + j] > 250) // Pixel is edge
		{
			for (int theta = 0; theta < 180; theta++)
			{
				float rho = (((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
				atomicAdd(&accumulators[(int)((round(rho + hough_h) * 180.0)) + theta], 1);
			}
		}
	}
}

__global__ void get_lines_kernel(int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, 
	float *sin_table, float *cos_table, int *x1_lines, int *y1_lines, int *x2_lines, int *y2_lines, int *lines)
{
	int rho = blockIdx.x * blockDim.x + threadIdx.x;
    int theta = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t max;

    if (rho < accu_height && theta < accu_width)
    {
        if (accumulators[(rho * accu_width) + theta] >= threshold)
        {
            // Is this point a local maxima (9x9)
            max = accumulators[(rho * accu_width) + theta];
            for (int ii = -4; ii <= 4; ii++)
			{
				for (int jj = -4; jj <= 4; jj++)
                {
                    if ((ii + rho >= 0 && ii + rho < accu_height) && (jj + theta >= 0 && jj + theta < accu_width))
                    {
                        if (accumulators[((rho + ii) * accu_width) + (theta + jj)] > max)
                        {
                            max = accumulators[((rho + ii) * accu_width) + (theta + jj)];
                        }
                    }
                }
			}
			
            if (max == accumulators[(rho * accu_width) + theta]) // local maxima
            {
                int x1, y1, x2, y2;
                x1 = y1 = x2 = y2 = 0;

                if (theta >= 45 && theta <= 135)
                {
                    if (theta > 90)
                    {
                        // y = (r - x cos(t)) / sin(t)
                        x1 = width / 2;
                        y1 = ((float)(rho - (accu_height / 2)) - ((x1 - (width / 2)) * cos_table[theta])) / sin_table[theta] + (height / 2);
                        x2 = width;
                        y2 = ((float)(rho - (accu_height / 2)) - ((x2 - (width / 2)) * cos_table[theta])) / sin_table[theta] + (height / 2);
                    }
                    else
                    {
                        // y = (r - x cos(t)) / sin(t)
                        x1 = 0;
                        y1 = ((float)(rho - (accu_height / 2)) - ((x1 - (width / 2)) * cos_table[theta])) / sin_table[theta] + (height / 2);
                        x2 = width * 2 / 5;
                        y2 = ((float)(rho - (accu_height / 2)) - ((x2 - (width / 2)) * cos_table[theta])) / sin_table[theta] + (height / 2);
                    }
                }
                else
                {
                    // x = (r - y sin(t)) / cos(t);
                    y1 = 0;
                    x1 = ((float)(rho - (accu_height / 2)) - ((y1 - (height / 2)) * sin_table[theta])) / cos_table[theta] + (width / 2);
                    y2 = height;
                    x2 = ((float)(rho - (accu_height / 2)) - ((y2 - (height / 2)) * sin_table[theta])) / cos_table[theta] + (width / 2);
                }

                int idx = atomicAdd(lines, 1);
                x1_lines[idx] = x1;
                y1_lines[idx] = y1;
                x2_lines[idx] = x2;
                y2_lines[idx] = y2;
            }
        }
    }
}

__global__ void draw_lines_kernel(uint8_t *imgtmp, int width, int height, int *x1, int *y1, int *x2, int *y2, int nlines, int width_line)
{
	int l = blockIdx.x * blockDim.x + threadIdx.x;
    int wl = blockIdx.y * blockDim.y + threadIdx.y;

    if (l < nlines && wl >= -(width_line >> 1) && wl <= (width_line >> 1))
    {	
        for (int x = x1[l]; x < x2[l]; x++)
        {
            int y = (float)(y2[l] - y1[l]) / (x2[l] - x1[l]) * (x - x1[l]) + y1[l]; // Line eq. known two points
            if (x + wl > 0 && x + wl < width && y > 0 && y < height)
            {
                imgtmp[3 * ((y) * width + x + wl)    ] = 255;
                imgtmp[3 * ((y) * width + x + wl) + 1] = 0;
                imgtmp[3 * ((y) * width + x + wl) + 2] = 0;
            }
        }
    }
}

void lane_assist_GPU(uint8_t *imgtmp, int height, int width, 
					 uint8_t *imEdge, int accu_height, int accu_width, 
					 int *x1, int *y1, int *x2, int *y2, int *nlines)
{
	// Device variables 
	float *d_sin_table, *d_cos_table;
	int *d_x1, *d_y1, *d_x2, *d_y2, *d_nlines;
	uint8_t *d_image_in, *d_image_out, *d_imEdge;
    float *d_NR, *d_G, *d_phi, *d_Gx, *d_Gy;
    uint8_t *d_pedge;
    uint32_t *d_accum;

	hipMalloc((void**)&d_sin_table, 	180 * 					   sizeof(float));
	hipMalloc((void**)&d_cos_table, 	180 * 					   sizeof(float));
	hipMalloc((void**)&d_x1, 			10  * 					   sizeof(int));
	hipMalloc((void**)&d_y1, 			10  * 					   sizeof(int));
	hipMalloc((void**)&d_x2, 			10  * 					   sizeof(int));
	hipMalloc((void**)&d_y2, 			10  * 					   sizeof(int));
	hipMalloc((void**)&d_nlines, 								   sizeof(int));
	hipMalloc((void**)&d_image_in,	 	3 * height * width * 	   sizeof(uint8_t));
	hipMalloc((void**)&d_image_out,	height * width * 		   sizeof(uint8_t));
    hipMalloc((void**)&d_imEdge, 		height * width * 		   sizeof(uint8_t));
    hipMalloc((void**)&d_NR, 			height * width * 		   sizeof(float));
    hipMalloc((void**)&d_G, 			height * width * 		   sizeof(float));
    hipMalloc((void**)&d_phi, 			height * width * 		   sizeof(float));
    hipMalloc((void**)&d_Gx, 			height * width * 		   sizeof(float));
    hipMalloc((void**)&d_Gy, 			height * width * 		   sizeof(float));
    hipMalloc((void**)&d_pedge, 		height * width * 		   sizeof(uint8_t));
    hipMalloc((void**)&d_accum, 		accu_width * accu_height * sizeof(uint32_t));

	// Initialize nlines to 0
	hipMemset(d_nlines, 0, sizeof(int));
	// Initialize accumulators to 0
	hipMemset(d_accum, 0, accu_width * accu_height * sizeof(uint32_t));
	
	// Copy image data to device
    hipMemcpy(d_image_in, imgtmp, 3 * height * width * sizeof(uint8_t), hipMemcpyHostToDevice);

	// Initialize sin and cos tables
	dim3 dimBlock(BLOCK_SIZE);
	int blocks = 180/BLOCK_SIZE;
	if (180%BLOCK_SIZE>0) blocks++;
	dim3 dimGrid(blocks);
    init_cos_sin_table_kernel<<<dimGrid, dimBlock>>>(d_sin_table, d_cos_table, 180);

	dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dimGrid = dim3(ceil((float) width / dimBlock.x), ceil((float) height / dimBlock.y));
	image_RGB2BW_kernel<<<dimGrid, dimBlock>>>(d_image_in, d_image_out, height, width);
	
	// Canny (edge detection)
	// Split canny_kernel into 5 kernels
	int level = 1000.0f;
	noiseReduction_kernel<<<dimGrid, dimBlock>>>(d_image_out, d_NR, height, width);
	gradient_kernel		 <<<dimGrid, dimBlock>>>(d_NR, d_G, d_phi, d_Gx, d_Gy, height, width);
	edge_kernel			 <<<dimGrid, dimBlock>>>(d_G, d_phi, d_pedge, height, width);
	thresholding_kernel	 <<<dimGrid, dimBlock>>>(d_image_out, d_imEdge, d_G, d_pedge, level, height, width);
	
	// Copy edge image to host
	hipMemcpy(imEdge, d_imEdge, height * width * sizeof(uint8_t), hipMemcpyDeviceToHost);

	// Hough transform (line detection)
	float hough_h = ((sqrt(2.0) * (float)(height > width ? height : width)) / 2.0);
    float center_x = width 	/ 2.0;
    float center_y = height / 2.0;
    hough_kernel<<<dimGrid, dimBlock>>>(d_imEdge, width, height, 
												   d_accum, accu_width, accu_height,
												   d_sin_table, d_cos_table, 
												   hough_h, center_x, center_y);
	
	int threshold;
	if (width > height) threshold = width / 6;
	else				threshold = height / 6;

    dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dimGrid = dim3(ceil((float) accu_height / dimBlock.x), ceil((float) accu_width / dimBlock.y));
    get_lines_kernel<<<dimGrid, dimBlock>>>(threshold, d_accum, 
		accu_width, accu_height, 
		width, height, 
		d_sin_table, d_cos_table, 
		d_x1, d_y1, d_x2, d_y2, d_nlines);
	
    // Copy results back to host
    hipMemcpy(x1, 		d_x1, 10 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(y1, 		d_y1, 10 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(x2, 		d_x2, 10 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(y2, 		d_y2, 10 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(nlines, 	d_nlines,  sizeof(int), hipMemcpyDeviceToHost);
	
	int width_line = 9;
    dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    dimGrid = dim3(ceil((float) *nlines / dimBlock.x), ceil((float) width_line / dimBlock.y));
	draw_lines_kernel<<<dimGrid, dimBlock>>>(d_image_in, width, height,
		d_x1, d_y1, d_x2, d_y2, *nlines, width_line);
    
	hipMemcpy(imgtmp, d_image_in, 3 * height * width * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(d_sin_table);
	hipFree(d_cos_table);
	hipFree(d_x1);
	hipFree(d_y1);
	hipFree(d_x2);
	hipFree(d_y2);
	hipFree(d_nlines);
	hipFree(d_image_in);
	hipFree(d_image_out);
    hipFree(d_imEdge);
    hipFree(d_NR);
    hipFree(d_G);
    hipFree(d_phi);
    hipFree(d_Gx);
    hipFree(d_Gy);
    hipFree(d_pedge);
    hipFree(d_accum);
}
